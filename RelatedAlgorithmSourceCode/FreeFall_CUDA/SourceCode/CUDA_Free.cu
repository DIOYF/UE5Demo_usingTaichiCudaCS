#include "hip/hip_runtime.h"
#pragma once

#include "CUDA_Free.h"
#include<iostream>

// CUDA RUNTIME API
__device__ float3 Vel[THREAD_NUM];
__device__ float3 Pos[THREAD_NUM];
__device__ float Range;
__device__ float ParticleRadius;
__device__ float Stiffness;
__device__ float RestitutionCoef;
__device__ float delta_t;
__global__ void MainCompute();


__device__ float3 RangeChecker(float3 pos, float3 vel)
{
    float3 velocity = vel;
    float range = Range / 2;
    // direction x
    if (pos.x >= range && velocity.x > 0)
    {
        velocity.x *= -0.6;
    }
    else if (pos.x <= -range && velocity.x < 0)
    {
        velocity.x *= -0.6;
    }
    // direction y
    if (pos.y >= range && velocity.y > 0)
    {
        velocity.y *= -0.6;
    }
    else if (pos.y <= -range && velocity.y < 0)
    {
        velocity.y *= -0.6;
    }
    // direction z
    if (pos.z >= range && velocity.z > 0)
    {
        velocity.z *= -0.6;
    }
    else if (pos.z <= -range && velocity.z < 0)
    {
        velocity.z *= -0.6;
    }
    return velocity;
}


__device__ float3 getAcceleration(int index, float3 position, float3 velocity)
{
    float K = Stiffness;
    float C = 2. * (1. / sqrt(1. + pow(3.14 / log(RestitutionCoef), 2))) * sqrt(K / 2);
    float collisionDistance = 2 * ParticleRadius;
    float3 acc_x = { 0.f, 0.f, 0.f };
    for (int i = 0; i < THREAD_NUM; i++)
    {
        if (index != i)
        {
            float3 neighborPos = Pos[i];
            float3 neighborVel = Vel[i];
            float3 rel_pos = { neighborPos.x - position.x, neighborPos.y - position.y, neighborPos.z - position.z };
            float distance = sqrt(pow(rel_pos.x, 2) + pow(rel_pos.y, 2) + pow(rel_pos.z, 2));
            if (distance < collisionDistance)
            {
                float3 normal = { rel_pos.x / distance, rel_pos.y/distance, rel_pos.z/distance};
                float3 f1 = { normal.x * (collisionDistance - distance) * K, normal.y * (collisionDistance - distance) * K, normal.z * (collisionDistance - distance) * K};
                float V = (neighborVel.x - velocity.x * normal.x)+(neighborVel.y - velocity.y * normal.y)+(neighborVel.z - velocity.z * normal.z);
                float3 f2 = { C * V * normal.x, C * V * normal.y, C * V * normal.z };
                acc_x = {acc_x.x + f2.x-f1.x, acc_x.y + f2.y-f1.y, acc_x.z + f2.z-f1.z};
            }
        }
    }
    return acc_x;
}


__global__ void MainCompute()
{
    int i = threadIdx.x;
    float3 gravity = {0., 0., -9.8};
    Vel[i].x = Vel[i].x + delta_t * getAcceleration(i, Pos[i], Vel[i]).x + delta_t * 1e4 * gravity.x;
    Vel[i].y = Vel[i].y + delta_t * getAcceleration(i, Pos[i], Vel[i]).y + delta_t * 1e4 * gravity.y;
    Vel[i].z = Vel[i].z + delta_t * getAcceleration(i, Pos[i], Vel[i]).z + delta_t * 1e4 * gravity.z;
    Vel[i] = RangeChecker(Pos[i], Vel[i]);

    Pos[i].x = Pos[i].x + delta_t * Vel[i].x;
    Pos[i].y = Pos[i].y + delta_t * Vel[i].y;
    Pos[i].z = Pos[i].z + delta_t * Vel[i].z;
}


// Compiled Lib Function Using For FreeFall Compute
hipError_t FreeFallCompute(metadata myMetaData, float3* positions, float3* velocities, std::string* error_message)
{
    float RangeValue = myMetaData.Range;
    float ParticleRadiusValue = myMetaData.ParticleRadius;
    float StiffnessValue = myMetaData.Stiffness;
    float RestitutionCoefValue = myMetaData.RestitutionCoef;
    float delta_tValue = myMetaData.delta_t;

    hipError_t cuda_status;
    // Choose GPU Device 0
    cuda_status = hipSetDevice(0);
    if (cuda_status != hipSuccess) {
        *error_message = "hipSetDevice failed!";
        goto Error;
    }

    // Transfer Global Data from CPU to GPU
    cuda_status = hipMemcpyToSymbol(HIP_SYMBOL(Vel), velocities, sizeof(float3) * THREAD_NUM);
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpyToSymbol Vel failed!";
        goto Error;
    }
    cuda_status = hipMemcpyToSymbol(HIP_SYMBOL(Pos), positions, sizeof(float3) * THREAD_NUM);
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpyToSymbol Pos failed!";
        goto Error;
    }
    cuda_status = hipMemcpyToSymbol(HIP_SYMBOL(Range), &RangeValue, sizeof(float));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpyToSymbol Range failed!";
        goto Error;
    }
    cuda_status = hipMemcpyToSymbol(HIP_SYMBOL(ParticleRadius), &ParticleRadiusValue, sizeof(float));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpyToSymbol ParticleRadius failed!";
        goto Error;
    }
    cuda_status = hipMemcpyToSymbol(HIP_SYMBOL(Stiffness), &StiffnessValue, sizeof(float));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpyToSymbol Stiffness failed!";
        goto Error;
    }
    cuda_status = hipMemcpyToSymbol(HIP_SYMBOL(RestitutionCoef), &RestitutionCoefValue, sizeof(float));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpyToSymbol RestitutionCoef failed!";
        goto Error;
    }
    cuda_status = hipMemcpyToSymbol(HIP_SYMBOL(delta_t), &delta_tValue, sizeof(float));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpyToSymbol delta_t failed!";
        goto Error;
    }

    // Using CUDA Kernel to Compute
    MainCompute << <1, THREAD_NUM, 1 >> > ();

    // Transfer Global Data from GPU to CPU
    cuda_status = hipMemcpyFromSymbol(velocities, HIP_SYMBOL(Vel), sizeof(float3) * THREAD_NUM);
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpyFromSymbol Vel failed!";
        goto Error;
    }
    cuda_status = hipMemcpyFromSymbol(positions, HIP_SYMBOL(Pos), sizeof(float3) * THREAD_NUM);
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpyFromSymbol Pos failed!";
        goto Error;
    }
    cuda_status = hipMemcpyFromSymbol(&RangeValue, HIP_SYMBOL(Range), sizeof(float));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpyFromSymbol Range failed!";
        goto Error;
    }
    cuda_status = hipMemcpyFromSymbol(&ParticleRadiusValue, HIP_SYMBOL(ParticleRadius), sizeof(float));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpyFromSymbol ParticleRadius failed!";
        goto Error;
    }
    cuda_status = hipMemcpyFromSymbol(&StiffnessValue, HIP_SYMBOL(Stiffness), sizeof(float));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpyFromSymbol Stiffness failed!";
        goto Error;
    }
    cuda_status = hipMemcpyFromSymbol(&RestitutionCoefValue, HIP_SYMBOL(RestitutionCoef), sizeof(float));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpyFromSymbol RestitutionCoef failed!";
        goto Error;
    }
    cuda_status = hipMemcpyFromSymbol(&delta_tValue, HIP_SYMBOL(delta_t), sizeof(float));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpyFromSymbol delta_t failed!";
        goto Error;
    }

Error:
    hipFree(Vel);
    hipFree(Pos);
    hipFree(&Range);
    hipFree(&ParticleRadius);
    hipFree(&Stiffness);
    hipFree(&RestitutionCoef);
    hipFree(&delta_t);

    return cuda_status;
}





