#include "hip/hip_runtime.h"
﻿#pragma once

#include "CudaClsi.h"
//#include "hip/device_functions.h"
#include <iostream>

// CUDA RUNTIME API
__device__ float3 Vel[THREAD_NUM_X * THREAD_NUM_Y];
__device__ float3 Pos[THREAD_NUM_X * THREAD_NUM_Y];
__device__ float spring_K;
__device__ float kinetic_damping;
__device__ float drag_damping;
__device__ float delta_t_1;
__device__ float3 wind_force;


__global__ void mainCompute_1();
//__device__ float3 Acc[THREAD_NUM_X * THREAD_NUM_Y];


__device__ float3 add_acceleration(int thread_x, int thread_y,float3 position, float3 velocity)
{
    float quad_size = 1.0f / 128.0f;
    float3 acc = { 0.f,0.f, -9.8f };
    for (int i = -1; i <= 1; i++)
    {
        for (int j = -1; j <= 1; j++)
        {
            if (i == 0 && j == 0)
            {
                continue;
            }
            else
            {
                int x = thread_x + i;
                int y = thread_y + j;
                if (0 <= x && x< THREAD_NUM_X && 0 <= y && y < THREAD_NUM_Y)
                {
                    float3 relative_pos = { position.x - Pos[x * THREAD_NUM_Y + y].x, position.y - Pos[x * THREAD_NUM_Y + y].y, position.z - Pos[x * THREAD_NUM_Y + y].z };
                    float3 relative_vel = { velocity.x - Vel[x * THREAD_NUM_Y + y].x, velocity.y - Vel[x * THREAD_NUM_Y + y].y, velocity.z - Vel[x * THREAD_NUM_Y + y].z };
                    //float3 direction = normal(relative_pos);

                    float current_distance = sqrtf( powf(relative_pos.x, 2) + powf(relative_pos.y, 2) + powf(relative_pos.z, 2) );
                    float original_distance = quad_size * sqrtf( powf(i * 1.0f,2) + powf(j * 1.0f,2));
                    

                    float3 direction = {relative_pos.x/current_distance, relative_pos.y/current_distance, relative_pos.z/current_distance};  

                    float K = relative_vel.x * direction.x + relative_vel.y * direction.y + relative_vel.z * direction.z;
                    
                    float relative_distance = (current_distance / original_distance - 1);

                    acc.x += -(spring_K  * relative_distance) * direction.x;
                    acc.x += -K * direction.x * kinetic_damping * quad_size;
                    
                    acc.y += -(spring_K * relative_distance)  * direction.y;
                    acc.y += -K * direction.y * kinetic_damping * quad_size;
                    
                    acc.z += -(spring_K * relative_distance)  * direction.z;
                    acc.z += -K * direction.z * kinetic_damping * quad_size;
                }
            }
        }
    }
    //if (isnan(acc.x) || isnan(acc.y) || isnan(acc.z)) return  { 0.f,0.f,-9.8f };
    return acc;
}


__global__ void mainCompute_1()
{
    int i = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y;
    //for (int i = 0; i < 20; i++)
    {
        float3 acc = add_acceleration(i, j, Pos[i * THREAD_NUM_Y + j], Vel[i * THREAD_NUM_Y + j]);

        //__threadfence();

        if (j != 0)
        {
            //float3 acc = Acc[i * THREAD_NUM_Y + j];
            Vel[i * THREAD_NUM_Y + j].x +=  (wind_force.x + acc.x) * delta_t_1;
            Vel[i * THREAD_NUM_Y + j].y +=  (wind_force.y + acc.y) * delta_t_1;
            Vel[i * THREAD_NUM_Y + j].z +=  (wind_force.z + acc.z) * delta_t_1;

            Vel[i * THREAD_NUM_Y + j].x *= exp(-drag_damping * delta_t_1);
            Vel[i * THREAD_NUM_Y + j].y *= exp(-drag_damping * delta_t_1);
            Vel[i * THREAD_NUM_Y + j].z *= exp(-drag_damping * delta_t_1);

            Pos[i * THREAD_NUM_Y + j].x += Vel[i * THREAD_NUM_Y + j].x * delta_t_1;
            Pos[i * THREAD_NUM_Y + j].y += Vel[i * THREAD_NUM_Y + j].y * delta_t_1;
            Pos[i * THREAD_NUM_Y + j].z += Vel[i * THREAD_NUM_Y + j].z * delta_t_1;
        }
    }
}


__global__ void mainCompute_2()
{
    int i = blockIdx.x * blockDim.x + threadIdx.x, j = blockIdx.y * blockDim.y + threadIdx.y;

}


// Compiled Lib Function Using for freefall compute
hipError_t ClothSimulationCompute(metadata myMetaData, float3* positions, float3* velocities, std::string* error_message)
{
    float spring_K_value = myMetaData.spring_K;
    float kinetic_damping_value = myMetaData.kinetic_damping;
    float drag_damping_value = myMetaData.drag_damping;
    float delta_t_value = myMetaData.delta_t / THREAD_NUM_X;
    float3 wind_force_value = { myMetaData.wind_strength * 0.f, myMetaData.wind_strength * 0.f, myMetaData.wind_strength * 0.f };
    
    hipError_t cuda_status;
    cuda_status = hipSetDevice(0);
    if (cuda_status != hipSuccess) {
        *error_message = "hipSetDevice failed!";
        goto Error;
    }

    cuda_status = hipMemcpyToSymbol(HIP_SYMBOL(Vel), velocities, sizeof(float3) * THREAD_NUM_X * THREAD_NUM_Y);
    if (cuda_status != hipSuccess) {
        *error_message = "cudaMemcpyToSymbol1 failed!";
        goto Error;
    }
    cuda_status = hipMemcpyToSymbol(HIP_SYMBOL(Pos), positions, sizeof(float3) * THREAD_NUM_X * THREAD_NUM_Y);
    if (cuda_status != hipSuccess) {
        *error_message = "cudaMemcpyToSymbol2 failed!";
        goto Error;
    }
    cuda_status = hipMemcpyToSymbol(HIP_SYMBOL(spring_K), &spring_K_value, sizeof(float));
    if (cuda_status != hipSuccess) {
        *error_message = "cudaMemcpyToSymbol3 failed!";
        goto Error;
    }
    cuda_status = hipMemcpyToSymbol(HIP_SYMBOL(kinetic_damping), &kinetic_damping_value, sizeof(float));
    if (cuda_status != hipSuccess) {
        *error_message = "cudaMemcpyToSymbol4 failed!";
        goto Error;
    }
    cuda_status = hipMemcpyToSymbol(HIP_SYMBOL(drag_damping), &drag_damping_value, sizeof(float));
    if (cuda_status != hipSuccess) {
        *error_message = "cudaMemcpyToSymbol5 failed!";
        goto Error;
    }
    cuda_status = hipMemcpyToSymbol(HIP_SYMBOL(delta_t_1), &delta_t_value, sizeof(float));
    if (cuda_status != hipSuccess) {
        *error_message = "cudaMemcpyToSymbol6 failed!";
        goto Error;
    }
    cuda_status = hipMemcpyToSymbol(HIP_SYMBOL(wind_force), &wind_force_value, sizeof(float3));
    if (cuda_status != hipSuccess) {
        *error_message = "cudaMemcpyToSymbol7 failed!";
        goto Error;
    }

    dim3 threadPerBlock = {32,32,1};
    dim3 blockPerGrid ={128/threadPerBlock.x, 128/threadPerBlock.y,1};
    // How to align Thread is what we need consider
    //MainCompute << < blockPerGrid, threadPerBlock >> > ();

    mainCompute_1 <<< blockPerGrid, threadPerBlock , 3 >> > ();

    cuda_status = hipMemcpyFromSymbol(velocities, HIP_SYMBOL(Vel), sizeof(float3) * THREAD_NUM_X * THREAD_NUM_Y);
    if (cuda_status != hipSuccess) {
        *error_message = "cudaMemcpyFromSymbol1 failed!";
        goto Error;
    }
    cuda_status = hipMemcpyFromSymbol(positions, HIP_SYMBOL(Pos), sizeof(float3) * THREAD_NUM_X * THREAD_NUM_Y);
    if (cuda_status != hipSuccess) {
        *error_message = "cudaMemcpyFromSymbol2 failed!";
        goto Error;
    }
    cuda_status = hipMemcpyFromSymbol(&spring_K_value, HIP_SYMBOL(spring_K), sizeof(float));
    if (cuda_status != hipSuccess) {
        *error_message = "cudaMemcpyFromSymbol3 failed!";
        goto Error;
    }
    cuda_status = hipMemcpyFromSymbol(&kinetic_damping_value, HIP_SYMBOL(kinetic_damping), sizeof(float));
    if (cuda_status != hipSuccess) {
        *error_message = "cudaMemcpyFromSymbol4 failed!";
        goto Error;
    }
    cuda_status = hipMemcpyFromSymbol(&drag_damping_value, HIP_SYMBOL(drag_damping),  sizeof(float));
    if (cuda_status != hipSuccess) {
        *error_message = "cudaMemcpyFromSymbol5 failed!";
        goto Error;
    }
    cuda_status = hipMemcpyFromSymbol(&drag_damping_value, HIP_SYMBOL(delta_t_1),  sizeof(float));
    if (cuda_status != hipSuccess) {
        *error_message = "cudaMemcpyFromSymbol6 failed!";
        goto Error;
    }
    cuda_status = hipMemcpyFromSymbol(&wind_force_value, HIP_SYMBOL(wind_force),  sizeof(float3));
    if (cuda_status != hipSuccess) {
        *error_message = "cudaMemcpyFromSymbol7 failed!";
        goto Error;
    }


Error:
    hipFree(Vel);
    hipFree(Pos);
    hipFree(&spring_K);
    hipFree(&kinetic_damping);
    hipFree(&drag_damping);
    hipFree(&delta_t_1);
    hipFree(&wind_force);

    return cuda_status;
}
